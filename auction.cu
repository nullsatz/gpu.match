#include "hip/hip_runtime.h"
#include<iostream>
#include"auction.h"

#define NTHREADS 16 // 256

// 2D float texture
texture<float, hipTextureType2D, hipReadModeElementType> d_benefits;

// each thread: an object
// for that object, look for the highest bid from unassigned person
__global__ void AuctionGPU_Assignment(int * d_numAssign, const int n,
	int * I, int * O, float * bids, float * p)
{
	int j = blockDim.x * blockIdx.x + threadIdx.x;
	if(j >= n) return;

	float
		tempBid = -1.0, highestBid = 0.0;
	int bidPerson = -1;
	
	//loop over people
	for(int i = 0; i < n; i++) {
		tempBid = bids[i * n + j];
		if(tempBid > highestBid) {
				highestBid = tempBid;
				bidPerson = i;
		}
	}

	if(bidPerson < 0) return;

	// the object j reviews the bid only if
	// bid person != currently assigned person
	if(O[j] == bidPerson) return;

	//unassign the person that was previously assigned to j:
	if(O[j] >= 0) I[O[j]] = -1;

	//raise the price to the winning bid
	//bidsRow = (float *) ((char *)bids + bidPerson * bidsPitch);
	//p[j] = bidsRow[j];
	p[j] = highestBid;

	//assign j to i
	I[bidPerson] = j;
	O[j] = bidPerson;
}

// each unassigned person i finds object j that offers max value to bid on
// each person may bid on any object
__global__ void AuctionGPU_Bidding(const int n, float * bids, float * p,
	int * I, const float  e, int * d_numAssign)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i >= n) return;
	if(I[i] != -1) return; //unassigned?
	
	//has Unassigned person 
	*d_numAssign = 1;

	// init the 2nd max object value with very low value
	// for the case when the person is only interested in one object
	int
		fir_maxObj = 0;
	float
		sec_maxObjValue = -1000.0, temp_ObjValue = 0.0;

	// float fir_maxObjValue = a[i * n] - p[0];
	float fir_maxObjValue = tex2D(d_benefits, 0, i) - p[0];
	for(int j = 1; j < n; j++) {
		temp_ObjValue = tex2D(d_benefits, j, i) - p[j];
		
		//if is higher that the highest
		if(temp_ObjValue > fir_maxObjValue) {
			sec_maxObjValue = fir_maxObjValue;

			fir_maxObj = j;
			fir_maxObjValue = temp_ObjValue;
		} else if(temp_ObjValue > sec_maxObjValue) {
			//or if is higher that the second highest
			sec_maxObjValue = temp_ObjValue;
		}
	}
	// bidding inc from person i for favorite object
	bids[i * n + fir_maxObj] = fir_maxObjValue - sec_maxObjValue + e;
}

void cudaTimerStart(hipEvent_t &start) {
	hipEventCreate(&start);
	hipEventRecord(start, 0 );
}

float cudaTimerEnd(hipEvent_t &start) {
	hipEvent_t stop;
	hipEventCreate(&stop);
	hipEventRecord( stop, 0 );
	hipEventSynchronize( stop );

	float time;
	hipEventElapsedTime( &time, start, stop );
	hipEventDestroy( start );
	hipEventDestroy( stop );
	return time;
}

int * d_auction(int cSize, float * h_a) {
	float e = 1.0;

	// Allocating CPU memory 
	// 'h_' prefix - CPU (host) memory space:

	// a[i,j] : desire of person i for object j
	// float * h_a = 0;		
	// h_a = (float *) malloc(sizeof(float) * C_MAX_INSTANCE * C_MAX_INSTANCE);

	// Allocating GPU memory
	// 'd_' prefix - GPU (device) memory space

	// Pick which CUDA capable device to run on
	hipSetDevice(0);
	// currently set to 0 which would be the default dive
	// if hipSetDevice weren't called at all

	hipEvent_t start;
	cudaTimerStart(start);

	//a [i,j] : desire of person i for object j
	// float * d_a;
	// hipMalloc((void **) & d_a, sizeof(float) * cSize * cSize);

	// Allocate CUDA array in device memory
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0,
		hipChannelFormatKindFloat);
	hipArray * cuArray;
	hipMallocArray(&cuArray, &channelDesc, cSize, cSize);

	// Copy to device memory some data located at address h_data
	// in host memory 
	hipMemcpyToArray(cuArray, 0, 0, h_a, sizeof(float) * cSize * cSize,
		hipMemcpyHostToDevice);

	// Set texture reference parameters
	d_benefits.addressMode[0] = hipAddressModeClamp;
	d_benefits.addressMode[1] = hipAddressModeClamp;
	d_benefits.filterMode = hipFilterModePoint;
	d_benefits.normalized = false;

	// Bind the array to the texture reference
	hipBindTextureToArray(d_benefits, cuArray, channelDesc);

	//bids value
	float * d_bids;
	hipMalloc(&d_bids, sizeof(float) * cSize * cSize);

	//p[j] : each object j has a price:
	float * d_p;
	hipMalloc((void **) & d_p, sizeof(float) * cSize);

	//each person is or not assigned
	int * d_i;
	hipMalloc((void **) & d_i, sizeof(int) * cSize);

	//each object is or not assigned
	int * d_o;
	hipMalloc((void **) & d_o, sizeof(int) * cSize);

	// used as a boolean that is set whenever there is an unassigned person
	int * d_numAssign;
	hipMalloc((void **) & d_numAssign, sizeof(int));
	
	dim3 dimBlock(NTHREADS, 1, 1);
	int gx = ceil(cSize /(double) dimBlock.x);
	dim3 dimGrid(gx, 1, 1);

	// copying input data to GPU mem and cleaning aux arrays.
	// in the case of the matrix could go mem constant:
	// so need to clear or copy the size you will use in interaction
	// hipMemcpy(d_a, h_a, sizeof(float) * cSize * cSize,
	// hipMemcpyHostToDevice);
	
	// cleaning/initializing algoritm mem
	hipMemset(d_bids, 0, cSize * cSize * sizeof(float));
	hipMemset(d_p, 0, cSize * sizeof(float));
	hipMemset(d_i, -1, cSize * sizeof(int));
	hipMemset(d_o, -1, cSize * sizeof(int));
	hipMemset(d_numAssign, 0, sizeof(int));

	int * h_numAssign;
	hipHostMalloc((void **) & h_numAssign, sizeof(int));
	*h_numAssign = 1;

	while(*h_numAssign > 0) {
		hipMemset(d_bids, 0, cSize * cSize * sizeof(float));
		hipMemset(d_numAssign, 0, sizeof(int));

		AuctionGPU_Bidding<<<dimBlock, dimGrid>>>(cSize, d_bids, d_p, d_i, e,
			d_numAssign);
            
		hipMemcpy(h_numAssign, d_numAssign, sizeof(int),
			hipMemcpyDeviceToHost);

		if(*h_numAssign > 0) {
			AuctionGPU_Assignment<<<dimBlock, dimGrid>>>(d_numAssign, cSize,
				d_i, d_o, d_bids, d_p);							
		}
	}
	hipHostFree(h_numAssign);
	hipFree(d_numAssign);

	// Release GPU memory
    // hipFree(d_a);
	hipFreeArray(cuArray);
    hipFree(d_bids);
    hipFree(d_p);
	hipFree(d_o);

	// person assignment results (contain the object number or -1 if unassigned)
	int * h_i_GPUresults = (int *) malloc(sizeof(int) * cSize);

	// Read back GPU results: read the assignements from d_I
	hipMemcpy(h_i_GPUresults, d_i, sizeof(int) * cSize,
		hipMemcpyDeviceToHost);
 
	hipFree(d_i);

	float time = cudaTimerEnd(start);
	std::cout << cSize << " " << time << std::endl;

	return h_i_GPUresults;
}
